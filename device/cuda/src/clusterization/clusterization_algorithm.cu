#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// CUDA Library include(s).
#include "../utils/utils.hpp"
#include "traccc/cuda/clusterization/clusterization_algorithm.hpp"
#include "traccc/cuda/utils/definitions.hpp"

// Project include(s)
#include "traccc/clusterization/device/connect_components.hpp"
#include "traccc/clusterization/device/count_cluster_cells.hpp"
#include "traccc/clusterization/device/create_measurements.hpp"
#include "traccc/clusterization/device/find_clusters.hpp"
#include "traccc/clusterization/device/form_spacepoints.hpp"
#include "traccc/cuda/utils/make_prefix_sum_buff.hpp"
#include "traccc/device/fill_prefix_sum.hpp"

// Vecmem include(s).
#include <vecmem/utils/copy.hpp>

// System include(s).
#include <algorithm>

std::size_t cellcount;
using scalar = TRACCC_CUSTOM_SCALARTYPE;
namespace traccc::cuda {
namespace kernels {

__global__ void fill_buffers(const cell_container_types::const_view cells_view,
                             vecmem::data::vector_view<unsigned int> channel0,
                             vecmem::data::vector_view<unsigned int> channel1,
vecmem::data::vector_view<scalar> activat,
                             vecmem::data::vector_view<unsigned int> cumulsize,
                             vecmem::data::vector_view<unsigned int> moduleidx) {

    cell_container_types::const_device cells_device(cells_view);
    vecmem::device_vector<unsigned int> ch0(channel0);
    vecmem::device_vector<unsigned int> ch1(channel1);
    vecmem::device_vector<scalar> activation(activat);
    vecmem::device_vector<unsigned int> sum(cumulsize);
    vecmem::device_vector<unsigned int> midx(moduleidx);

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= cells_device.size())
        return;

    const auto& cells = cells_device.at(idx).items;
    
    unsigned int doffset = 0;
    for (int i=0; i < idx; i++) {
        doffset+= cells_device.at(i).items.size();
    }
    sum.at(idx) = doffset;

    if (idx == cells_device.size() - 1) {
        sum.at(idx+1) = doffset + cells_device.at(idx).items.size();
    }

    std::size_t n_cells = cells.size();
    for (int i=0; i < n_cells; i++) {
        ch0.at(i+doffset) = cells[i].channel0;
        ch1.at(i+doffset) = cells[i].channel1;
activation.at(i+doffset)=cells[i].activation;
        midx.at(i+doffset) = idx;
    }
}

__global__ void find_clusters(
    const cell_container_types::const_view cells_view,
    vecmem::data::vector_view<unsigned int> channel0,
    vecmem::data::vector_view<unsigned int> channel1,
    vecmem::data::vector_view<unsigned int> cumulsize,
    vecmem::data::vector_view<unsigned int> moduleidx,
    vecmem::data::vector_view<unsigned int> label_view,
    vecmem::data::vector_view<std::size_t> clusters_per_module_view) {

    device::find_clusters(threadIdx.x + blockIdx.x * blockDim.x, cells_view,
                          channel0, channel1, cumulsize, moduleidx,
                          label_view, clusters_per_module_view);
}

__global__ void fill2(vecmem::data::vector_view<unsigned int> label_view,
                      vecmem::data::jagged_vector_view<unsigned int> sparse_ccl_indices_view,
                      vecmem::data::vector_view<unsigned int> cumulsize) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= cumulsize.size()-1)
        return;
    
    vecmem::device_vector<unsigned int> labels(label_view);
    vecmem::jagged_device_vector<unsigned int> device_sparse_ccl_indices(
        sparse_ccl_indices_view);
    vecmem::device_vector<unsigned int> sum(cumulsize);

    unsigned int doffset = sum[idx];
    const unsigned int n_cells = sum[idx+1] - doffset;
    for (int i=0; i < n_cells; i++) {
        device_sparse_ccl_indices[idx][i] = labels[i+doffset];
    }
}

 __global__ void fill3(const cell_container_types::const_view cells_view,
    vecmem::data::vector_view<unsigned int > Clusters_module_link ,
    vecmem::data::vector_view<point2 > measurement_local,
    vecmem::data::vector_view<variance2 > measurement_variance,
    measurement_container_types::view measurements_view )
    {
       int idx = threadIdx.x + blockIdx.x * blockDim.x;
       if (idx >= Clusters_module_link.size())
         return;
    cell_container_types::const_device cells_device(cells_view);
    vecmem::device_vector<unsigned int> Cl_module_link(Clusters_module_link);
    vecmem::device_vector<point2> local_measurement(measurement_local);
    vecmem::device_vector<variance2> variance_measurement(measurement_variance);
    measurement_container_types::device measurements_device(measurements_view);
    
    std::size_t module_link_ = Cl_module_link[idx];
    point2 local_ = local_measurement[idx];
    variance2 variance_ = variance_measurement[idx];
    measurement m;
    m.cluster_link = module_link_;
    m.local = local_;
    m.variance = variance_;
    auto &module = cells_device.at(module_link_).header;
    measurements_device[module_link_].header = module;
    measurements_device[module_link_].items.push_back(std::move(m));
    }

__global__ void count_cluster_cells(
    vecmem::data::vector_view<unsigned int> label_view,
    vecmem::data::vector_view<std::size_t> cluster_prefix_sum_view,
     vecmem::data::vector_view<unsigned int> moduleidx,
   vecmem::data::vector_view<unsigned int> cells_cl_prefix_sum,
    vecmem::data::vector_view<unsigned int> cluster_sizes_view) {

    device::count_cluster_cells(
        threadIdx.x + blockIdx.x * blockDim.x, label_view,
        cluster_prefix_sum_view,moduleidx, cells_cl_prefix_sum, cluster_sizes_view);
}

__global__ void connect_components(
     vecmem::data::vector_view<unsigned int> moduleidx,
     vecmem::data::vector_view<unsigned int> label_view,
     vecmem::data::vector_view<std::size_t> cluster_prefix_sum_view,
     vecmem::data::vector_view<unsigned int> cluster_idx_atomic,
     vecmem::data::vector_view<unsigned int> cells_cl_prefix_sum,
    vecmem::data::vector_view<unsigned int> clusters_view) {

    device::connect_components(threadIdx.x + blockIdx.x * blockDim.x,
                               moduleidx, label_view,
                               cluster_prefix_sum_view, cluster_idx_atomic,cells_cl_prefix_sum,
                               clusters_view, 0);
}

__global__ void create_measurements(
    vecmem::data::vector_view<unsigned int > moduleidx,
    vecmem::data::vector_view<scalar> activation_cell,
    vecmem::data::vector_view<unsigned int> channel0,
    vecmem::data::vector_view<unsigned int> channel1,
    vecmem::data::vector_view<unsigned int > clusters_view,
    vecmem::data::vector_view<unsigned int > cel_cl_ps, // cell_cluster_prefix_sum
    const cell_container_types::const_view cells_view,
    measurement_container_types::view measurements_view ,
    vecmem::data::vector_view<unsigned int > Clusters_module_link ,
    vecmem::data::vector_view<point2 > measurement_local,
    vecmem::data::vector_view<variance2 > measurement_variance) {

    device::create_measurements(threadIdx.x + blockIdx.x * blockDim.x,
                              moduleidx ,activation_cell,channel0, channel1,
                                clusters_view,cel_cl_ps, cells_view,
                                 measurements_view,Clusters_module_link, measurement_local, measurement_variance);
}

__global__ void form_spacepoints(
    measurement_container_types::const_view measurements_view,
    vecmem::data::vector_view<const device::prefix_sum_element_t>
        measurements_prefix_sum_view,
    spacepoint_container_types::view spacepoints_view) {

    device::form_spacepoints(threadIdx.x + blockIdx.x * blockDim.x,
                             measurements_view, measurements_prefix_sum_view,
                             spacepoints_view);
}

}  // namespace kernels

clusterization_algorithm::clusterization_algorithm(
    const traccc::memory_resource& mr, vecmem::copy& copy, stream& str)
    : m_mr(mr), m_copy(copy), m_stream(str) {}

clusterization_algorithm::output_type clusterization_algorithm::operator()(
    const cell_container_types::const_view& cells_view) const {

    // Get a convenience variable for the stream that we'll be using.
    hipStream_t stream = details::get_stream(m_stream);

    // Number of modules
    const cell_container_types::const_device::header_vector::size_type
        num_modules = m_copy.get_size(cells_view.headers);

    // Work block size for kernel execution
    const std::size_t threadsPerBlock = 64;

    // Get the sizes of the cells in each module
    const std::vector<
        cell_container_types::const_device::item_vector::value_type::size_type>
        cell_sizes = m_copy.get_sizes(cells_view.items);

    cellcount = 0;
    for (int i=0; i < cell_sizes.size(); i++) {
        cellcount += cell_sizes[i];
    }

    //cellvec cells;
    vecmem::data::vector_buffer<unsigned int> channel0(cellcount, m_mr.main);
    m_copy.setup(channel0);
    vecmem::data::vector_buffer<unsigned int> channel1(cellcount, m_mr.main);
    m_copy.setup(channel1);
    vecmem::data::vector_buffer<scalar> activation(cellcount, m_mr.main);
    m_copy.setup(activation);
    vecmem::data::vector_buffer<unsigned int> moduleidx(cellcount, m_mr.main);
    m_copy.setup(moduleidx);
    vecmem::data::vector_buffer<unsigned int> prefixsum(num_modules+1, m_mr.main);
    m_copy.setup(prefixsum);

    std::size_t blocksPerGrid = (num_modules + threadsPerBlock - 1) / threadsPerBlock;
    kernels::fill_buffers<<<blocksPerGrid, threadsPerBlock, 0, stream>>>
                            (cells_view, channel0, channel1,activation, prefixsum, moduleidx);

    /*
     * Helper container for sparse CCL calculations.
     * Each inner vector corresponds to 1 module.
     * The indices in a particular inner vector will be filled by sparse ccl
     * and will indicate to which cluster, a particular cell in the module
     * belongs to.
     */
    /*vecmem::data::jagged_vector_buffer<unsigned int> sparse_ccl_indices_buff(
        std::vector<std::size_t>(cell_sizes.begin(), cell_sizes.end()),
        m_mr.main, m_mr.host);
    m_copy.setup(sparse_ccl_indices_buff);*/

    vecmem::data::vector_buffer<unsigned int> label_buff(cellcount, m_mr.main);
    m_copy.setup(label_buff);

    /*
     * cl_per_module_prefix_buff is a vector buffer with numbers of found
     * clusters in each module. Later it will be transformed into prefix sum
     * vector (hence the name). The logic is the following. After
     * cluster_finding_kernel, the buffer will contain cluster sizes e.i.
     *
     * cluster sizes: | 1 | 12 | 5 | 102 | 42 | ... - cl_per_module_prefix_buff
     * module index:  | 0 |  1 | 2 |  3  |  4 | ...
     *
     * Now, we copy those cluster sizes to the host and make a duplicate vector
     * of them. So, we are left with cl_per_module_prefix_host, and
     * clusters_per_module_host - which are the same. Now, we procede to
     * modifying the cl_per_module_prefix_host to actually resemble its name
     * i.e.
     *
     * We do std::inclusive_scan on it, which will result in a prefix sum
     * vector:
     *
     * cl_per_module_prefix_host: | 1 | 13 | 18 | 120 | 162 | ...
     *
     * Then, we copy this vector into the previous cl_per_module_prefix_buff.
     * In this way, we don't need to allocate the memory on the device twice.
     *
     * Now, the monotonic prefix sum buffer - cl_per_module_prefix_buff, will
     * allow us to insert the clusters at the correct position inside the
     * kernel. The remaining host vector - clusters_per_module_host, will be
     * needed to allocate memory for other buffers later in the code.
     */
    vecmem::data::vector_buffer<std::size_t> cl_per_module_prefix_buff(
        num_modules, m_mr.main);
    m_copy.setup(cl_per_module_prefix_buff);

    // Calculating grid size for cluster finding kernel
    blocksPerGrid =
        (num_modules + threadsPerBlock - 1) / threadsPerBlock;

    // Invoke find clusters that will call cluster finding kernel
    kernels::find_clusters<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        cells_view, channel0, channel1, prefixsum, moduleidx,
        label_buff, cl_per_module_prefix_buff);
    CUDA_ERROR_CHECK(hipGetLastError());

    /*kernels::fill2<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        label_buff, sparse_ccl_indices_buff, prefixsum);
    CUDA_ERROR_CHECK(hipGetLastError());*/

    // Create prefix sum buffer
    vecmem::data::vector_buffer cells_prefix_sum_buff =
        make_prefix_sum_buff(cell_sizes, m_copy, m_mr, m_stream);

    // Copy the sizes of clusters per module to the host
    // and create a copy of "clusters per module" vector
    vecmem::vector<std::size_t> cl_per_module_prefix_host(
        m_mr.host ? m_mr.host : &(m_mr.main));
    m_copy(cl_per_module_prefix_buff, cl_per_module_prefix_host,
           vecmem::copy::type::copy_type::device_to_host);
    m_stream.synchronize();
    std::vector<std::size_t> clusters_per_module_host(
        cl_per_module_prefix_host.begin(), cl_per_module_prefix_host.end());

    // Perform the inclusive scan operation
    std::inclusive_scan(cl_per_module_prefix_host.begin(),
                        cl_per_module_prefix_host.end(),
                        cl_per_module_prefix_host.begin());

    unsigned int total_clusters = cl_per_module_prefix_host.back();

    // Copy the prefix sum back to its device container
    m_copy(vecmem::get_data(cl_per_module_prefix_host),
           cl_per_module_prefix_buff,
           vecmem::copy::type::copy_type::host_to_device);

    // Vector of the exact cluster sizes, will be filled in cluster counting
    vecmem::data::vector_buffer<unsigned int> cluster_sizes_buffer(
        total_clusters, m_mr.main);
    m_copy.setup(cluster_sizes_buffer);
    m_copy.memset(cluster_sizes_buffer, 0);

    printf("capacity : %llu\n", cells_prefix_sum_buff.capacity());
    // Calclating grid size for cluster counting kernel (block size 64)
    blocksPerGrid = (cells_prefix_sum_buff.capacity() + threadsPerBlock - 1) /
                    threadsPerBlock;
    // Invoke cluster counting will call count cluster cells kernel
    vecmem::data::vector_buffer<unsigned int> cells_cluster_ps(total_clusters, m_mr.main);
    m_copy.setup(cells_cluster_ps);//prefix sum cells per cluster 
    kernels::count_cluster_cells<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        label_buff, cl_per_module_prefix_buff, moduleidx, cells_cluster_ps,
        cluster_sizes_buffer);
    // Check for kernel launch errors and Wait for the cluster_counting kernel
    // to finish
    CUDA_ERROR_CHECK(hipGetLastError());

    // Copy cluster sizes back to the host
    vecmem::vector<unsigned int> cluster_sizes{m_mr.host ? m_mr.host
                                                         : &(m_mr.main)};
    m_copy(cluster_sizes_buffer, cluster_sizes,
           vecmem::copy::type::copy_type::device_to_host);
    m_stream.synchronize();

    // Cluster container buffer for the clusters and headers (cluster ids)
    cluster_container_types::buffer clusters_buffer{
        {total_clusters, m_mr.main},
        {std::vector<std::size_t>(total_clusters, 0),
         std::vector<std::size_t>(cluster_sizes.begin(), cluster_sizes.end()),
         m_mr.main, m_mr.host}};
    m_copy.setup(clusters_buffer.headers);
    m_copy.setup(clusters_buffer.items);

    vecmem::data::vector_buffer<unsigned int> cluster_index_atomic(total_clusters, m_mr.main);
    m_copy.setup(cluster_index_atomic);
    m_copy.memset(cluster_index_atomic, 0);
    vecmem::data::vector_buffer<unsigned int> clusters_buff(cellcount, m_mr.main);
    m_copy.setup(clusters_buff);

    // Using previous block size and thread size (64)
    // Invoke connect components will call connect components kernel
    kernels::connect_components<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        moduleidx, label_buff, cl_per_module_prefix_buff, cluster_index_atomic,
        cells_cluster_ps, clusters_buff);
    CUDA_ERROR_CHECK(hipGetLastError());

    // Resizable buffer for the measurements
    measurement_container_types::buffer measurements_buffer{
        {num_modules, m_mr.main},
        {std::vector<std::size_t>(num_modules, 0), clusters_per_module_host,
         m_mr.main, m_mr.host}};
    m_copy.setup(measurements_buffer.headers);
    m_copy.setup(measurements_buffer.items);

    // Spacepoint container buffer to fill inside the spacepoint formation
    // kernel
    spacepoint_container_types::buffer spacepoints_buffer{
        {num_modules, m_mr.main},
        {std::vector<std::size_t>(num_modules, 0), clusters_per_module_host,
         m_mr.main, m_mr.host}};
    m_copy.setup(spacepoints_buffer.headers);
    m_copy.setup(spacepoints_buffer.items);

    printf("clusters_buff %llu\n", clusters_buffer.headers.size());
    // Calculating grid size for measurements creation kernel (block size 64)
    blocksPerGrid = (clusters_buffer.headers.size() - 1 + threadsPerBlock) /
                    threadsPerBlock;
    
    
    //measurement struct 
    vecmem::data::vector_buffer<unsigned int> Clusters_module_link(total_clusters, m_mr.main);
    m_copy.setup(Clusters_module_link);
    m_copy.memset(Clusters_module_link, 0);

    vecmem::data::vector_buffer<point2> measurement_local(total_clusters, m_mr.main);
    m_copy.setup(measurement_local);
    //m_copy.memset(measurement_local, 0);

    vecmem::data::vector_buffer<variance2> measurement_variance(total_clusters, m_mr.main);
    m_copy.setup(measurement_variance);
    //m_copy.memset(measurement_variance, 0);


    // Invoke measurements creation will call create measurements kernel
    kernels::create_measurements<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        moduleidx, activation ,channel0, channel1,clusters_buff,cells_cluster_ps,cells_view,
        measurements_buffer, Clusters_module_link,measurement_local, measurement_variance);
    CUDA_ERROR_CHECK(hipGetLastError());
   
   
   //kernel fill 3 
   
    measurement_container_types::buffer measurement_buff{
        {num_modules, m_mr.main},
        {std::vector<std::size_t>(num_modules, 0), clusters_per_module_host,
         m_mr.main, m_mr.host}};
    m_copy.setup(measurement_buff.headers);
    m_copy.setup(measurement_buff.items);

    kernels::fill3<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
       cells_view, Clusters_module_link,measurement_local, measurement_variance,measurement_buff );
    // Create prefix sum buffer
    vecmem::data::vector_buffer meas_prefix_sum_buff = make_prefix_sum_buff(
        std::vector<device::prefix_sum_size_t>{clusters_per_module_host.begin(),
                                               clusters_per_module_host.end()},
        m_copy, m_mr, m_stream);

    // Using the same grid size as before
    // Invoke spacepoint formation will call form_spacepoints kernel
    kernels::form_spacepoints<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        measurement_buff, meas_prefix_sum_buff, spacepoints_buffer);
    CUDA_ERROR_CHECK(hipGetLastError());

    // Return the buffer. Which may very well not be filled at this point yet.
    return spacepoints_buffer;
}

}  // namespace traccc::cuda
