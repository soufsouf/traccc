#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021-2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "traccc/cuda/seeding/spacepoint_binning.hpp"
#include "traccc/cuda/utils/definitions.hpp"

// Project include(s).
#include "traccc/seeding/device/count_grid_capacities.hpp"
#include "traccc/seeding/device/populate_grid.hpp"

// VecMem include(s).
#include <vecmem/utils/copy.hpp>
#include <vecmem/utils/cuda/copy.hpp>

namespace traccc::cuda {
namespace kernels {

/// CUDA kernel for running @c traccc::device::count_grid_capacities
__global__ void count_grid_capacities(
    seedfinder_config config, sp_grid::axis_p0_type phi_axis,
    sp_grid::axis_p1_type z_axis,
    spacepoint_collection_types::const_view spacepoints,
    vecmem::data::vector_view<unsigned int> grid_capacities) {

    device::count_grid_capacities(threadIdx.x + blockIdx.x * blockDim.x, config,
                                  phi_axis, z_axis, spacepoints,
                                  grid_capacities);
}
__global__ void count_grid_capacities2(
    seedfinder_config config, sp_grid::axis_p0_type phi_axis,
    sp_grid::axis_p1_type z_axis,
    const traccc::spacepoint_collection_types::const_view spacepoints_view,
    unsigned int size,
    vecmem::data::vector_view<unsigned int> grid_capacities) {

    device::count_grid_capacities2(threadIdx.x + blockIdx.x * blockDim.x, config,
                                  phi_axis, z_axis, spacepoints_view, size,
                                  grid_capacities);
}

/// CUDA kernel for running @c traccc::device::populate_grid
__global__ void populate_grid(
    seedfinder_config config,
    spacepoint_collection_types::const_view spacepoints, sp_grid_view grid) {

    device::populate_grid(threadIdx.x + blockIdx.x * blockDim.x, config,
                          spacepoints, grid);
}
__global__ void populate_grid2(
    seedfinder_config config,
    const traccc::spacepoint_collection_types::const_view spacepoints_view,
    unsigned int size,
    sp_grid_view grid) {

    device::populate_grid2(threadIdx.x + blockIdx.x * blockDim.x, config,
                          spacepoints_view, size, grid);
}
}  // namespace kernels

spacepoint_binning::spacepoint_binning(
    const seedfinder_config& config, const spacepoint_grid_config& grid_config,
    const traccc::memory_resource& mr)
    : m_config(config.toInternalUnits()),
      m_axes(get_axes(grid_config.toInternalUnits(),
                      (mr.host ? *(mr.host) : mr.main))),
      m_mr(mr) {

    // Initialize m_copy ptr based on memory resources that were given
    if (mr.host) {
        m_copy = std::make_unique<vecmem::cuda::copy>();
    } else {
        m_copy = std::make_unique<vecmem::copy>();
    }
}

sp_grid_buffer spacepoint_binning::operator()(
    const spacepoint_collection_types::const_view& spacepoints_view) const {

    // Get the spacepoint sizes from the view
    auto sp_size = m_copy->get_size(spacepoints_view);

    // Set up the container that will be filled with the required capacities for
    // the spacepoint grid.
    const std::size_t grid_bins = m_axes.first.n_bins * m_axes.second.n_bins;
    vecmem::data::vector_buffer<unsigned int> grid_capacities_buff(grid_bins,
                                                                   m_mr.main);
    m_copy->setup(grid_capacities_buff);
    m_copy->memset(grid_capacities_buff, 0);
    vecmem::data::vector_view<unsigned int> grid_capacities_view =
        grid_capacities_buff;

    // Calculate the number of threads and thread blocks to run the kernels for.
    const unsigned int num_threads = WARP_SIZE * 8;
    const unsigned int num_blocks = (sp_size + num_threads - 1) / num_threads;

    // Fill the grid capacity container.
    kernels::count_grid_capacities<<<num_blocks, num_threads>>>(
        m_config, m_axes.first, m_axes.second, spacepoints_view,
        grid_capacities_view);
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    // Copy grid capacities back to the host
    vecmem::vector<unsigned int> grid_capacities_host(m_mr.host ? m_mr.host
                                                                : &(m_mr.main));
    (*m_copy)(grid_capacities_buff, grid_capacities_host);

    // Create the grid buffer.
    sp_grid_buffer grid_buffer(
        m_axes.first, m_axes.second, std::vector<std::size_t>(grid_bins, 0),
        std::vector<std::size_t>(grid_capacities_host.begin(),
                                 grid_capacities_host.end()),
        m_mr.main, m_mr.host);
    m_copy->setup(grid_buffer._buffer);
    sp_grid_view grid_view = grid_buffer;

    // Populate the grid.
    kernels::populate_grid<<<num_blocks, num_threads>>>(
        m_config, spacepoints_view, grid_view);
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    // Return the freshly filled buffer.
    return grid_buffer;
}


spacepoint_binning2::spacepoint_binning2(
    const seedfinder_config& config, const spacepoint_grid_config& grid_config,
    const traccc::memory_resource& mr)
    : m_config(config.toInternalUnits()),
      m_axes(get_axes(grid_config.toInternalUnits(),
                      (mr.host ? *(mr.host) : mr.main))),
      m_mr(mr) {

    // Initialize m_copy ptr based on memory resources that were given
    if (mr.host) {
        m_copy = std::make_unique<vecmem::cuda::copy>();
    } else {
        m_copy = std::make_unique<vecmem::copy>();
    }
}

    sp_grid_buffer spacepoint_binning2::operator()(
    const spacepoint_collection_types::const_view& spacepoints_view, const unsigned int &spacepoints_size) const {

    // Get the spacepoint sizes from the view
    auto sp_size = spacepoints_size;

    // Set up the container that will be filled with the required capacities for
    // the spacepoint grid.
    const std::size_t grid_bins = m_axes.first.n_bins * m_axes.second.n_bins;
    vecmem::data::vector_buffer<unsigned int> grid_capacities_buff(grid_bins,
                                                                   m_mr.main);
    m_copy->setup(grid_capacities_buff);
    m_copy->memset(grid_capacities_buff, 0);
    vecmem::data::vector_view<unsigned int> grid_capacities_view =
        grid_capacities_buff;

    // Calculate the number of threads and thread blocks to run the kernels for.
    const unsigned int num_threads = WARP_SIZE * 8;
    const unsigned int num_blocks = (sp_size + num_threads - 1) / num_threads;

    // Fill the grid capacity container.
    kernels::count_grid_capacities2<<<num_blocks, num_threads>>>(
        m_config, m_axes.first, m_axes.second, spacepoints_view, sp_size,
        grid_capacities_view);
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    // Copy grid capacities back to the host
    vecmem::vector<unsigned int> grid_capacities_host(m_mr.host ? m_mr.host
                                                                : &(m_mr.main));
    (*m_copy)(grid_capacities_buff, grid_capacities_host);

    // Create the grid buffer.
    sp_grid_buffer grid_buffer(
        m_axes.first, m_axes.second, std::vector<std::size_t>(grid_bins, 0),
        std::vector<std::size_t>(grid_capacities_host.begin(),
                                 grid_capacities_host.end()),
        m_mr.main, m_mr.host);
    m_copy->setup(grid_buffer._buffer);
    sp_grid_view grid_view = grid_buffer;

    // Populate the grid.
    kernels::populate_grid2<<<num_blocks, num_threads>>>(
        m_config, spacepoints_view, sp_size, grid_view);
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    // Return the freshly filled buffer.
    return grid_buffer;
}
}  // namespace traccc::cuda
